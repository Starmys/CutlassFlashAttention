// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.


#include <vector>

#include <hip/hip_fp16.h>

#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>

#include "cutlass/cutlass.h"
#include "cutlass/layout/matrix.h"

#include "cutlass/epilogue/threadblock/epilogue_with_visitor.h"
#include "cutlass/fast_math.h"

#include "kernel_forward.h"
#include "kernel_backward.h"


// struct Arch {
//   static int const kMinComputeCapability = __CUDA_ARCH__ / 10; 
// };


std::vector<at::Tensor> fmha_forward(
  at::Tensor Q, // B, Nt, H, D
  at::Tensor K, // B, Ns, H, D
  at::Tensor V, // B, Ns, H, D
  float scale,
  bool calc_lse
) {
  hipSetDevice(Q.get_device());
  int B = Q.size(0);
  int Ns = K.size(1);
  int Nt = Q.size(1);
  int H = Q.size(2);
  int D = Q.size(3);
  auto opts = Q.options();
  at::Tensor O = torch::zeros_like(Q, opts);

  static constexpr int kMaxK = 64; // <- Decrease to 32/16 if your problem is smaller
  static int const kQueriesPerBlock = 64;
  static int const kKeysPerBlock = 64;

  using ForwardKernel = AttentionKernel<
    cutlass::half_t,      // scalar_t
    cutlass::arch::Sm80,  // ArchTag
    true,                 // Memory is aligned
    kQueriesPerBlock,
    kKeysPerBlock,
    kMaxK,
    false,                // Supports dropout
    false                 // Supports bias
  >;

  typename ForwardKernel::Params p;
  p.query_ptr = (cutlass::half_t*)(Q.data_ptr<c10::Half>());
  p.key_ptr = (cutlass::half_t*)(K.data_ptr<c10::Half>());
  p.value_ptr = (cutlass::half_t*)(V.data_ptr<c10::Half>());
  p.logsumexp_ptr = nullptr;
  p.output_accum_ptr = nullptr;
  if (ForwardKernel::kNeedsOutputAccumulatorBuffer) {
    hipMalloc(&p.output_accum_ptr, B * H * Nt * D * sizeof(typename ForwardKernel::output_accum_t));
  }
  p.output_ptr = (cutlass::half_t*)(O.data_ptr<c10::Half>());

  p.scale = scale;
  p.num_heads = H;
  p.num_batches = B;
  p.head_dim = D;
  p.head_dim_value = D;
  p.num_queries = Nt;
  p.num_keys = Ns;

  p.q_strideH = D;
  p.k_strideH = D;
  p.v_strideH = D;
  p.q_strideM = p.q_strideH * H;
  p.k_strideM = p.k_strideH * H;
  p.v_strideM = p.v_strideH * H;
  p.q_strideB = p.q_strideM * Nt;
  p.k_strideB = p.k_strideM * Ns;
  p.v_strideB = p.v_strideM * Ns;
  p.o_strideM = p.head_dim_value * p.num_heads;

  std::vector<at::Tensor> outputs = {O};
  if (calc_lse) {
    at::Tensor lse = torch::empty({B, H, Nt}, opts.dtype(at::kFloat));
    p.logsumexp_ptr = lse.data_ptr<float>();
    outputs.push_back(lse);
  }

  constexpr auto kernel_fn = attention_kernel_batched_impl<ForwardKernel>;

  int smem_bytes = sizeof(typename ForwardKernel::SharedStorage);
  if (smem_bytes > 0xc000) {
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
  }

  if (!ForwardKernel::check_supported(p)) {
    std::cerr << "Kernel does not support these inputs" << std::endl;
    return outputs;
  }

  kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes>>>(p);

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess)  {
    std::cerr << "Kernel execution error: " << hipGetErrorString(result);
  }

  return outputs;
}


std::vector<at::Tensor> fmha_backward(
  at::Tensor Q, // B, Nt, H, D
  at::Tensor K, // B, Ns, H, D
  at::Tensor V, // B, Ns, H, D
  at::Tensor O, // B, Nt, H, D
  at::Tensor dO, // B, Nt, H, D
  at::Tensor lse, // B, H, Nt
  at::Tensor delta, // B, H, Nt
  float scale
) {
  hipSetDevice(Q.get_device());
  int B = Q.size(0);
  int Ns = K.size(1);
  int Nt = Q.size(1);
  int H = Q.size(2);
  int D = Q.size(3);
  at::Tensor dQ = torch::empty_like(Q, Q.options());
  at::Tensor dK = torch::empty_like(K, K.options());
  at::Tensor dV = torch::empty_like(V, V.options());

  static constexpr int kMaxK = 64;
  static constexpr int kBlockSizeI = 64;
  static constexpr int kBlockSizeJ = 64;

  using BackwardKernel = AttentionBackwardKernel<
      cutlass::arch::Sm80,
      cutlass::half_t,
      true,        // kIsAligned_
      false,       // kApplyDropout_
      false,       // kPreload_
      kBlockSizeI, // kBlockSizeI_,
      kBlockSizeJ, // kBlockSizeJ_,
      kMaxK,       // kMaxK
      false,       // kKeysQueriesAlignedToBlockSize
      true         // kEnableSplitKeys
  >;

  typename BackwardKernel::Params p;
  p.query_ptr = (cutlass::half_t*)(Q.data_ptr<c10::Half>());
  p.key_ptr = (cutlass::half_t*)(K.data_ptr<c10::Half>());
  p.value_ptr = (cutlass::half_t*)(V.data_ptr<c10::Half>());
  p.output_ptr = (cutlass::half_t*)(O.data_ptr<c10::Half>());
  p.logsumexp_ptr = lse.data_ptr<float>();
  p.delta_ptr = delta.data_ptr<float>();
  p.grad_output_ptr = (cutlass::half_t*)(dO.data_ptr<c10::Half>());
  p.grad_query_ptr = (cutlass::half_t*)(dQ.data_ptr<c10::Half>());
  p.grad_key_ptr = (cutlass::half_t*)(dK.data_ptr<c10::Half>());
  p.grad_value_ptr = (cutlass::half_t*)(dV.data_ptr<c10::Half>());

  p.scale = scale;
  p.num_heads = H;
  p.num_batches = B;
  p.head_dim = D;
  p.head_dim_value = D;
  p.num_queries = Nt;
  p.num_keys = Ns;

  p.q_strideH = D;
  p.k_strideH = D;
  p.v_strideH = D;
  p.o_strideH = D;
  p.gQ_strideH = D;
  p.gK_strideH = D;
  p.gV_strideH = D;
  p.gO_strideH = D;

  p.q_strideM = p.q_strideH * H;
  p.k_strideM = p.k_strideH * H;
  p.v_strideM = p.v_strideH * H;
  p.gO_strideM = p.gO_strideH * H;

  p.gQKV_strideM_multiplier = 1;
  p.q_strideB = p.q_strideM * Nt;
  p.k_strideB = p.k_strideM * Ns;
  p.v_strideB = p.v_strideM * Ns;
  p.o_strideB = p.o_strideM() * Nt;
  p.gQ_strideB = p.gQ_strideM() * Nt;
  p.gK_strideB = p.gK_strideM() * Ns;
  p.gV_strideB = p.gV_strideM() * Ns;
  p.gO_strideB = p.gO_strideM * Nt;

  p.lse_strideH = Nt;
  p.delta_strideH = Nt;
  p.lse_strideB = p.lse_strideH * H;
  p.delta_strideB = p.delta_strideH * H;

  p.num_splits_key = Ns / 64;

  if (p.workspace_size()) {
      hipMalloc(&p.workspace, p.workspace_size());
  }

  std::vector<at::Tensor> outputs = {dQ, dK, dV};

  auto kernel_fn = attention_kernel_backward_batched_impl<BackwardKernel>;

  int smem_bytes = sizeof(typename BackwardKernel::SharedStorage);
  if (smem_bytes > 0xc000) {
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
  }

  if (!BackwardKernel::check_supported(p)) {
    std::cerr << "Kernel does not support these inputs" << std::endl;
    return outputs;
  }

  kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes>>>(p);

  // Wait for completion
  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess)  {
    std::cerr << "Kernel execution error: " << hipGetErrorString(result);
  }

  return outputs;
}
